
#include <hip/hip_runtime.h>
#ifndef M_PI
#define M_PI 3.1415926535897932384626433832795f
#endif

template<bool use_center_filter>
__device__ void update_f_value(float2 *f, float2 g0t, float x0, float y0,
                               float coeff0, float coeff1,
                               int center_half_size, int ell0, int ell1,
                               int stride, int n);

template<>
__device__ void update_f_value<false>(float2 *f, float2 g0, float x0, float y0,
                                      float coeff0, float coeff1,
                                      int center_half_size, int ell0, int ell1,
                                      int stride, int n)
{
  float w0 = ell0 / (float)(2 * n) - x0;
  float w1 = ell1 / (float)(2 * n) - y0;
  float w = coeff0 * __expf(coeff1 * (w0 * w0 + w1 * w1));
  float2 g0t = make_float2(w*g0.x, w*g0.y);
  int f_ind = ell0 + stride * ell1;
  atomicAdd(&(f[f_ind].x), g0t.x);
  atomicAdd(&(f[f_ind].y), g0t.y);
}

template<>
__device__ void update_f_value<true>(float2 *f, float2 g0, float x0, float y0,
                                     float coeff0, float coeff1,
                                     int center_half_size, int ell0, int ell1,
                                     int stride, int n)
{ 
  if( ell0 < -center_half_size || ell0 >= center_half_size ||
      ell1 < -center_half_size || ell1 >= center_half_size ) {      
    float w0 = ell0 / (float)(2 * n) - x0;
    float w1 = ell1 / (float)(2 * n) - y0;
    float w = coeff0 * __expf(coeff1 * (w0 * w0 + w1 * w1));
    float2 g0t = make_float2(w*g0.x, w*g0.y);
    int f_ind = ell0 + stride * ell1;
    atomicAdd(&(f[f_ind].x), g0t.x);
    atomicAdd(&(f[f_ind].y), g0t.y);
  }
}

template<bool use_center_filter>
__device__ void gather_kernel_common(float2 *g, float2 *f, float *theta, 
                                     int m, float mu, 
                                     int center_size, int n, int nproj, int nz)    
{
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;

  const int center_half_size = center_size/2;

  if (tx >= n || ty >= nproj || tz >= nz)
    return;
  float2 g0, g0t;
  float coeff0, coeff1;
  float x0, y0;
  int ell0, ell1, g_ind, f_ind;

  g_ind = tx + ty * n + tz * n * nproj;
  coeff0 = M_PI / mu;
  coeff1 = -M_PI * M_PI / mu;
  float sintheta, costheta;
  __sincosf(theta[ty], &sintheta, &costheta);
  x0 = (tx - n / 2) / (float)n * costheta;
  y0 = -(tx - n / 2) / (float)n * sintheta;
  if (x0 >= 0.5f)
    x0 = 0.5f - 1e-5;
  if (y0 >= 0.5f)
    y0 = 0.5f - 1e-5;

  int stride1 = 2*n + 2*m;
  int stride2 = stride1 * stride1;

  g0.x = g[g_ind].x;
  g0.y = g[g_ind].y;

  // offset f by [tz, n+m, n+m]
  f += n+m + (n+m) * stride1 + tz * stride2;
  
  #pragma unroll
  for (int i1 = 0; i1 < 2 * m + 1; i1++)
  {
    ell1 = floorf(2 * n * y0) - m + i1;
    #pragma unroll
    for (int i0 = 0; i0 < 2 * m + 1; i0++)
    {
      ell0 = floorf(2 * n * x0) - m + i0;
      update_f_value<use_center_filter>(f, g0, x0, y0, coeff0, coeff1, 
                                        center_half_size, 
                                        ell0, ell1, stride1, n);
    }
  }
}

extern "C" __global__ void gather_kernel_partial(float2 *g, float2 *f, float *theta, 
                                                 int m, float mu, 
                                                 int center_size, int n, int nproj, int nz)    
{
  gather_kernel_common<true>(g, f, theta, m, mu, center_size, n, nproj, nz);
}

extern "C" __global__ void gather_kernel(float2 *g, float2 *f, float *theta, 
                                         int m, float mu, int n, int nproj, int nz)    
{
  gather_kernel_common<false>(g, f, theta, m, mu, 0, n, nproj, nz);
}

/*m = 4
mu = 2.6356625556996645e-05
n = 362
nproj = 241
nz = 128
g (128, 241, 362)
f (128, 732, 732)
theta (241,)*/

bool __device__ compare(float *theta, int nproj, int index, float value, bool less) {
  if( index == 0 && value > theta[0] )
    return true;
  if( index == (nproj - 1)  && value < theta[(nproj - 1) ])
    return true;
  
  if(less) {
    if (theta[index - 1] > value && value >= theta[index])
      return true;
    else
      return false;
  } else {
    if (theta[index] > value && value >= theta[index + 1])
      return true;
    else
      return false;
  }
}

int __device__ binary_search(float *theta, int nproj, float value, bool less) {
    int low = 0, high = nproj - 1; 
    while (low <= high) {
        int middle = low + (high - low) / 2;

        if (compare(theta, nproj, middle, value, less))
            return middle;

        if (theta[middle] > value)
            low = middle + 1;
        else
            high = middle - 1;
    }

    return low;
}

extern "C" __global__ void gather_kernel_center_prune_v2(int* angle_range, float *theta,
                                                         int m, int center_size,
                                                         int n, int nproj)
{

  const int center_half_size = center_size/2;

  int thread_x = blockDim.x * blockIdx.x + threadIdx.x;
  int thread_y = blockDim.y * blockIdx.y + threadIdx.y;

  int tx = max(0, n + m - center_half_size) + thread_x;
  int ty = max(0, n + m - center_half_size) + thread_y; 

  if (thread_x >= center_size || thread_y >= center_size)
    return;

  int f_stride = 2*n + 2*m;
  int f_stride_2 = f_stride * f_stride;

  const float radius_2 =  2.f * (float(m) + 0.5f) * (float(m) + 0.5f) / f_stride_2;

  // offset angle_index_out by thread_x and thread_y
  angle_range += (unsigned long long)3 * (thread_x + thread_y * center_size);
  // Point coordinates
  float2 point   = make_float2(float(tx - (n+m)) / float(2 * n), float((n+m) - ty) / float(2 * n));
  float length_2 = point.x * point.x + point.y * point.y;

  if( radius_2 >= length_2 ) {
    angle_range[0] = 0;
    angle_range[1] = nproj - 1;
    angle_range[2] = 1;
  } else {
    double radius     = sqrt(radius_2);
    double length     = sqrt(length_2);
    double acosangle  = acos((double)point.x/length);
    double angle      = point.y > 0.f ? -(M_PI - acosangle) : -acosangle;
    float angle_delta = atan(radius/length);

    float angle_min = angle + angle_delta;
    float angle_max = angle - angle_delta;

    //float angle_range_delta = atan(radius/0.5f);

    //float angle_range_min = theta[nproj - 1]; // - fabsf(angle_range_delta);
    //float angle_range_max = theta[0]          + fabsf(angle_range_delta);

    if( fabsf(point.y) > radius ) {
    //if( abs(double((n+m) - ty) / double(2 * n)) > radius ) {

    //if( angle_range_min < angle_min && angle_min < angle_range_max &&
    //    angle_range_min < angle_max && angle_max < angle_range_max ) {
      angle_range[0] = binary_search(theta, nproj, angle_min, false);
      angle_range[1] = binary_search(theta, nproj, angle_max, true);
      angle_range[2] = 1;
    } else {
      angle_min = angle_min < -M_PI ? (angle_min + M_PI) : angle_min;
      angle_max = angle_max < -M_PI ? (angle_max + M_PI) : angle_max;

      angle_min = angle_min > 0 ? (angle_min - M_PI) : angle_min;
      angle_max = angle_max > 0 ? (angle_max - M_PI) : angle_max;

      int index_min = binary_search(theta, nproj, angle_min, true);
      int index_max = binary_search(theta, nproj, angle_max, false);
      if(index_min < index_max) {
        angle_range[0] = index_min;
        angle_range[1] = index_max;
      } else {
        angle_range[0] = index_max;
        angle_range[1] = index_min;
      }
      angle_range[2] = 0;
    }
  }
}

#define FULL_MASK 0xffffffff

extern "C" __global__ void gather_kernel_center_prune(int* angle_range, float *theta, 
                                                      int m, 
                                                      int center_size, 
                                                      int center_size_x, int center_size_y,
                                                      int n, int nproj)
{
  const int center_half_size = center_size/2;

  int thread_x = threadIdx.x;
  int thread_y = blockDim.y * blockIdx.y + threadIdx.y;
  int thread_z = blockDim.z * blockIdx.z + threadIdx.z;

  int tx = max(0, n + m - center_size_x / 2) + thread_y;
  int ty = max(0, n + m - center_size_y / 2) + thread_z; 

  if (thread_y >= center_size_x || thread_z >= center_size_y)
    return;

  int f_stride = 2*n + 2*m;
  int f_stride_2 = f_stride * f_stride;

  const float radius_2 =  2.f * (float(m) + 0.5f) * (float(m) + 0.5f) / f_stride_2;

  // offset angle_index_out by thread_x and thread_y
  angle_range += (unsigned long long)3 * (thread_y + (center_size - center_size_x)/2  + ((center_size - center_size_y)/2 + thread_z) * center_size);
  // Point coordinates
  float2 point = make_float2(float(tx - (n+m)) / float(2 * n), float((n+m) - ty) / float(2 * n));

  unsigned thread_mask = FULL_MASK >> (32 - thread_x);

  // Result value
  int valid_index = 0;
  int proj_valid_index_min = nproj;
  int proj_valid_index_max = 0;
  int proj_invalid_index_min = nproj;
  int proj_invalid_index_max = 0;
  int nproj_ceil = (nproj / 32 + 1) * 32;
  for (int proj_index = thread_x; proj_index < nproj_ceil; proj_index +=32) {
    float sintheta, costheta;
    __sincosf(theta[proj_index%nproj], &sintheta, &costheta);

    float polar_radius   = 0.5;
    float polar_radius_2 = polar_radius * polar_radius;

    float2 vector_polar = make_float2(polar_radius * costheta, polar_radius * sintheta);
    float2 vector_point = make_float2(point.x,  point.y);

    float dot = vector_polar.x * vector_point.x + vector_polar.y * vector_point.y;
    float2 mid_point = make_float2(dot * vector_polar.x / polar_radius_2, 
                                   dot * vector_polar.y / polar_radius_2); 

    float distance_2 = (mid_point.x - vector_point.x) * (mid_point.x - vector_point.x) +
                       (mid_point.y - vector_point.y) * (mid_point.y - vector_point.y);
  
    unsigned mask = __ballot_sync(FULL_MASK, radius_2 >= distance_2 && proj_index < nproj);
    
    if( proj_index < nproj ) {
      if(radius_2 >= distance_2) {
        proj_valid_index_min = min(proj_valid_index_min, proj_index);
        proj_valid_index_max = max(proj_valid_index_max, proj_index);
      } else {
        proj_invalid_index_min = min(proj_invalid_index_min, proj_index);
        proj_invalid_index_max = max(proj_invalid_index_max, proj_index);
      }
    }

    valid_index += __popc(mask);
  }

  // Find the minimum and maximum indices
  #pragma unroll
  for (int offset = 16; offset > 0; offset /= 2) {
    int proj_valid_index_min_temp = __shfl_down_sync(FULL_MASK, proj_valid_index_min, offset);
    proj_valid_index_min = min(proj_valid_index_min, proj_valid_index_min_temp);
    int proj_valid_index_max_temp = __shfl_down_sync(FULL_MASK, proj_valid_index_max, offset);
    proj_valid_index_max = max(proj_valid_index_max, proj_valid_index_max_temp);

    int proj_invalid_index_min_temp = __shfl_down_sync(FULL_MASK, proj_invalid_index_min, offset);
    proj_invalid_index_min = min(proj_invalid_index_min, proj_invalid_index_min_temp);
    int proj_invalid_index_max_temp = __shfl_down_sync(FULL_MASK, proj_invalid_index_max, offset);
    proj_invalid_index_max = max(proj_invalid_index_max, proj_invalid_index_max_temp);
  }

  if( thread_x == 0 ) {
    if((valid_index - 1) == (proj_valid_index_max - proj_valid_index_min)) {
      angle_range[0] = proj_valid_index_min;
      angle_range[1] = proj_valid_index_max;
      angle_range[2] = 1;
    } else {
      angle_range[0] = proj_invalid_index_min;
      angle_range[1] = proj_invalid_index_max;
      angle_range[2] = 0;
    }
  }
}

__device__ void inline 
gather_kernel_center_common(float2 *g, float *theta, 
                            float2& f_value, const float2& point,
                            const float& radius_2,
                            int proj_index, int tz,
                            const float coeff0,
                            const float coeff1,
                            int n, int nproj)    
{
  float sintheta, costheta;
  __sincosf(theta[proj_index], &sintheta, &costheta);

  float polar_radius   = 0.5;
  float polar_radius_2 = polar_radius * polar_radius;

  float2 vector_polar = make_float2(polar_radius * costheta, polar_radius * sintheta);
  float2 vector_point = make_float2(point.x, point.y);

  float dot = vector_polar.x * vector_point.x + vector_polar.y * vector_point.y;
  float2 mid_point = make_float2(dot * vector_polar.x / polar_radius_2, 
                                 dot * vector_polar.y / polar_radius_2); 

  float distance_2 = (mid_point.x - vector_point.x) * (mid_point.x - vector_point.x) +
                     (mid_point.y - vector_point.y) * (mid_point.y - vector_point.y);

  if( radius_2 >= distance_2 ) {
    
    // Distance to intersect
    float distance_to_intersect = sqrtf(radius_2 - distance_2);

    int radius_min, radius_max;
    if( fabsf(vector_polar.x) > fabsf(vector_polar.y) ) {
      radius_min = n/2 - 1 + floorf((mid_point.x - distance_to_intersect * vector_polar.x / polar_radius) / (2.f * vector_polar.x / n));
      radius_max = n/2 + 1 + floorf((mid_point.x + distance_to_intersect * vector_polar.x / polar_radius) / (2.f * vector_polar.x / n));
    } else {
      radius_min = n/2 - 1 + floorf((mid_point.y - distance_to_intersect * vector_polar.y / polar_radius) / (2.f * vector_polar.y / n));
      radius_max = n/2 + 1 + floorf((mid_point.y + distance_to_intersect * vector_polar.y / polar_radius) / (2.f * vector_polar.y / n));
    }

    if( radius_min > radius_max ) {
      int temp(radius_max); radius_max = radius_min; radius_min = temp;
    }

    radius_min = min( max(radius_min, 0), (n-1));
    radius_max = min( max(radius_max, 0), (n-1));

    constexpr int length = 4;
    float2 f_values[length];
    for (int radius_index = radius_min; radius_index < radius_max; radius_index+=length) {
      
      #pragma unroll
      for (int i = 0; i < length; i++) {
        int g_ind = radius_index + i + proj_index * n + tz * n * nproj;
        if( radius_index + i < radius_max ) {
          f_values[i].x = g[g_ind].x;
          f_values[i].y = g[g_ind].y;
        } else {
          f_values[i].x = 0.f;
          f_values[i].y = 0.f;
        }
      }

      #pragma unroll
      for (int i = 0; i < length; i++) {
        float x0 = (radius_index + i - n / 2) / (float)n * costheta;
        float y0 = (radius_index + i - n / 2) / (float)n * sintheta;

        if (x0 >= 0.5f)
          x0 = 0.5f - 1e-5;
        if (y0 >= 0.5f)
          y0 = 0.5f - 1e-5;

        float w0 = point.x - x0;
        float w1 = point.y - y0;
        float w = coeff0 * __expf(coeff1 * (w0 * w0 + w1 * w1));

        f_values[i].x *= w;
        f_values[i].y *= w;
      }

      #pragma unroll
      for (int i = 0; i < length; i++) {
        f_value.x += f_values[i].x;
        f_value.y += f_values[i].y;
      }
    }
  }
}

extern "C" __global__ void gather_kernel_center(float2 *g, float2 *f, 
                                                int* angle_range, float *theta, 
                                                int m, float mu,  
                                                int center_size,
                                                int n, int nproj, int nz)            
{

  const int center_half_size = center_size/2;

  int thread_x = blockDim.x * blockIdx.x + threadIdx.x;
  int thread_y = blockDim.y * blockIdx.y + threadIdx.y;
  int thread_z = blockDim.z * blockIdx.z + threadIdx.z;

  int tx = max(0, n + m - center_half_size) + thread_x;
  int ty = max(0, n + m - center_half_size) + thread_y; 
  int tz = thread_z;

  if (thread_x >= center_size || thread_y >= center_size || tz >= nz)
    return;

  const float coeff0 = M_PI / mu;
  const float coeff1 = -M_PI * M_PI / mu;

  int f_stride = 2*n + 2*m;
  int f_stride_2 = f_stride * f_stride;

  // offset f by tz
  f += (unsigned long long)tz * f_stride_2;
  // offset angle_index_out by thread_x and thread_y
  angle_range += (unsigned long long)3 * (thread_x + thread_y * center_size);

  const float radius_2 =  2.f * (float(m) + 0.5f) * (float(m) + 0.5f) / f_stride_2;

  // Result value
  float2 f_value = make_float2(0.f, 0.f);
  // Point coordinates
  float2 point = make_float2(float(tx - (n+m)) / float(2 * n), float((n+m) - ty) / float(2 * n));
  
  if( angle_range[2] ) {
    for (int proj_index = angle_range[0]; proj_index <= angle_range[1]; proj_index++) {
      gather_kernel_center_common(g, theta, 
                                  f_value, point,
                                  radius_2,
                                  proj_index, tz,
                                  coeff0,
                                  coeff1,
                                  n, nproj);
    }
  } else {
    for (int proj_index = 0; proj_index < angle_range[0]; proj_index++) {
      gather_kernel_center_common(g, theta, 
                                  f_value, point,
                                  radius_2,
                                  proj_index, tz,
                                  coeff0,
                                  coeff1,
                                  n, nproj);
    }
    for (int proj_index = angle_range[1] + 1; proj_index < nproj; proj_index++) {
      gather_kernel_center_common(g, theta, 
                                  f_value, point,
                                  radius_2,
                                  proj_index, tz,
                                  coeff0,
                                  coeff1,
                                  n, nproj);
    }
  }

  // index of the force
  int f_ind = tx + ty * f_stride;

  f[f_ind].x = f_value.x;
  f[f_ind].y = f_value.y;
}

extern "C" __global__ void wrap_kernel(float2 *f,
                                       int n, int nz, int m)
{
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int ty = blockDim.y * blockIdx.y + threadIdx.y;
  int tz = blockDim.z * blockIdx.z + threadIdx.z;

  if (tx >= 2 * n + 2 * m || ty >= 2 * n + 2 * m || tz >= nz)
    return;
  if (tx < m || tx >= 2 * n + m || ty < m || ty >= 2 * n + m)
  {
    int tx0 = (tx - m + 2 * n) % (2 * n);
    int ty0 = (ty - m + 2 * n) % (2 * n);
    int id1 = tx + ty * (2 * n + 2 * m) + tz * (2 * n + 2 * m) * (2 * n + 2 * m);
    int id2 = tx0 + m + (ty0 + m) * (2 * n + 2 * m) + tz * (2 * n + 2 * m) * (2 * n + 2 * m);

    atomicAdd(&f[id2].x, f[id1].x);
    atomicAdd(&f[id2].y, f[id1].y);
  }
}
